#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {
    const hipError_t error = call;
    if (error != cudaSucess) {
        printf("Error: %s:%d, ", __FILE__, __LINE__);
        printf("Code:%d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double eps = 1.0e-8;
    bool match = 1;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > eps) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if (match) printf("Arrays match.\n\n");
}

void initialize_data(float *ip, int size) {
    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF)/10.0f;
    }
}

void sumArraysHost(float *A, float *B, float *C, const int N) {
    for (int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArraysGPU(float *A, float *B, float *C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    C[idx] = A[idx] + B[idx];
}

int main(int argc, char** argv) {
    printf("%s Starting ...\n", argv[0]);

    // set up device
    int dev = 0;
    hipSetDevice(dev);

    // set up data size of vectors
    int nElem = 32;
    printf("Vector size %d\n", nELem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    // initialize data at host side
    initialize_data(h_A, nElem);
    initialize_data(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // invoke kernel
    dim3 block (nElem);
    dim3 grid (nElem/block.x);
    sumArraysGPU<<<grid, block>>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d, %d>>>\n", grid.x, block.x);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // add vector on host side
    sumArraysHost(h_A, h_B, hostRef, nElem);

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}